
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

using namespace std;

typedef struct celdas
{
    int filas;
    int columnas;
    double *elementos;
}celdas;

#define M2V(f, c, nf, mc) (f * mc + c)

double Uinicial(int );
double UinicialVel(int );

__global__ void KernelPrincipal(double *elem, double C, int t, int n, int m, int nGPU)
{
    // falta poner lo de multiGPU
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (( i >= 1 ) && ( i < m - 1 ))
    {
        // ecuación principal.
        elem[(t + 1) * m + i] = elem[t * m + i] + C * (elem[t * m + (i+1)] - 2*elem[t * m + i] + elem[t * m + (i - 1)]); 
    }
    if (i == 0 && t == 0) printf("Mugre t0: %lf ", elem[0]);
}

void imprimirMatriz(double *elem, int n, int m)
{
    for (int t = 0; t < n; t++)
    {
        for (int i = 0; i < m; i ++)
        {
            cout << elem[ t*m + i ] << ' '; // a ceros, para una prueba inicial
        }
    cout << endl;
    }
}

int main(int argc, char *argv[])
{
    int n = 64; // número de filas
    int m = 64; // número de columnas
    // lo escribo así porque después mejoraré el código
    if (n*m > 4194304)
    {
        cout << "Matriz demasiado grande." << endl;
        return 1;
    }
    celdas U, d_U, V;
    U.columnas = m;
    U.filas = n;
    size_t memSize = n*m*sizeof(double);
    U.elementos = (double*)malloc(memSize);
    d_U.columnas = m;
    d_U.filas = n;
    hipMalloc(&d_U.elementos,  memSize);
    V.columnas = m;
    V.filas = n;
    V.elementos = (double*)malloc(memSize);
    // condiciones iniciales y de frontera
    const double UbordeInicial = 100.0;
    const double UbordeFinal = 0.0;
    // propiedades físicas de la barra, acero
    const double k = 1;
    const double deltaT = 0.01; // cada décima de segundo 
    const double deltaX = 1; // la barra es m veces este valor
    const double C = ( k * deltaT ) / deltaX;
    // i, j, k para las dimensiones espaciales, t para el tiempo
    //  en este caso, i va con las x.
    for (int t = 0; t < n; t++)
    {
        for (int i = 0; i < m; i ++)
        {
            U.elementos[ t * m + i ] = 0.0; // a ceros, para una prueba inicial
        }
    }
    // condiciones iniciales y de frontera
    for (int i = 0; i < m; i ++)
    {
        U.elementos[ 0 * m + i ] = Uinicial(i);
    }
    for (int t = 0; t < n; t++)
    {
        U.elementos[ t * m + 0 ] = UbordeInicial;
        U.elementos[ t * m + (m - 1) ] = UbordeFinal;
    }
    cout << endl;
    imprimirMatriz(U.elementos, n, m);
    // copia a GPU
    hipMemcpy(d_U.elementos, U.elementos, memSize, hipMemcpyHostToDevice);
    // ciclo principal en cpu
    cout << "CPU:" << endl;
    int t, i;
    for (t = 0; t < n - 1; t++)
    {
        for (i = 1; i < m - 1; i++)
        {
            // ecuación principal.
            U.elementos[(t + 1) * m + i] = U.elementos[t * m + i] + C * (U.elementos[t * m + (i+1)] - 2*U.elementos[t * m + i] + U.elementos[t * m + (i - 1)]); 
        }
        // cout << "Paso de tiempo " << t << endl;
        cout << ".";
        //imprimirMatriz(U.elementos, n, m);
    }
    cout << endl;
    //imprimirMatriz(U.elementos, n, m);
    // ciclo principal en GPU
    cout << "GPU:" << endl;
    for (t = 0; t < n - 1; t++)
    {
        //la implementación de streams acá aparecerá en la medida en que se intercalen copias, o se divida en multiple GPU
        KernelPrincipal<<< n / 64, 64>>>(d_U.elementos, C, t, n, m, 1);
        hipDeviceSynchronize();
        // cout << "Paso de tiempo " << t << '\n';
        cout << ".";
        if ( t == 0 )
        {
            hipMemcpy(V.elementos, d_U.elementos, memSize, hipMemcpyDeviceToHost);
            cout << "paso t = 0 \n";
            imprimirMatriz(V.elementos, n, m);
        }
    }
    hipMemcpy( V.elementos, d_U.elementos, memSize, hipMemcpyDeviceToHost);
    //imprimirMatriz(V.elementos, n, m);
    for (i = 0; i < m; i++)
    {
        cout << U.elementos[ (n - 1) * m + i ] << " " << V.elementos[ (n - 1) * m + i ] << " ";
    }
}

double Uinicial(int i)
{
    // esto para facilitar cambiar las condiciones iniciales en el futuro
    return 0.0;
}

double UinicialVel(int i)
{
    // esto para facilitar cambiar las condiciones iniciales en el futuro
    return 0.0;
}
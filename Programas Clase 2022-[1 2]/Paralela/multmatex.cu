
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"
#include "math.h"
#include "time.h"

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)

typedef struct {
	int width;
	int height;
	double* elements;
} Matrix;

// thread block size
// #define BLOCK_SIZE 16
// #define BLOCK_SIZE 4

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C);

// Matrix multiplication - host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C) {
	// load A and B to devic memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(double);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	// allocate C in device memory 
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	printf("\n\n  dimensiones de C (antes del kernel): %d, %d\n",C.width,C.height);
	size = C.width * C.height * sizeof(double);
	hipMalloc(&d_C.elements, size);
	int BLOCK_SIZE = 16;
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	MatMulKernel<<<dimGrid,dimBlock>>>(d_A,d_B,d_C);
	// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	// porqueria
	int i,j;
	printf("\n\n  dimensiones de C: %d, %d\n",C.height,C.width);
	/*for (i=0; i< C.width ; ++i) 
	for (j=0; j < C.height; ++j) {			
		//printf("\n\n%f", C.elements[i*C.width+j]); 
	}*/
	
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
	// each thread compute one element of C
	// by acumulating results into Cvalue
	double Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < A.width; ++e) {
		Cvalue = Cvalue + A.elements[row *A.width + e] * B.elements[e * B.width + col];
	}
	C.elements[row * C.width + col] = Cvalue;
}

size_t index(int filas, int cols, int ancho)
{
	return (filas *  ancho + cols);
}

int main( int argc, char** argv) {
	// crear dos matrices guardadas en row major order, y una más para el resultado
	int filas = 1024;
	int cols;
	cols = filas;

	// double A_elem, B_elem, C_elem;
	Matrix A, B, C;

	double* A_elem = new double[filas * cols];
	double* B_elem = new double[filas * cols];
	double* C_elem = new double[filas * cols];
	double* C_elem_GPU = new double[filas * cols];
	
	// vamos al ejemplo de la matriz cuadrada, 8x8
	A.width = cols;
	B.width = cols;
	C.width = cols;
	A.height = filas;
	B.height = filas;
	C.height = filas;

	int count = 0;
	int i, j, k;
	printf("\n\n\n\n\n");
	printf("\n************************************\n");
	printf("Prueba de multiplicacion de matrices");
	printf("\n************************************\n");
	// llenado de la matriz A.
	for (i=0; i< A.width ; ++i)
	{
		for (j=0; j < A.height; ++j)
		{
			A_elem[index(i, j, cols)] = rand()%10*1.0;
			
		}
	}
	// llenado de la matriz B.
	for (i=0; i< B.width ; ++i) {
		for (j=0; j < B.height; ++j) {
			B_elem[index(i, j, cols)] = rand()%10*1.0;
		}
	}
	// Ahora la impresión inicial de las dos matrices

	// La A
	count = 0;
	printf("\n\n -- Matriz A --\n");
	for (i=0; i< A.width ; ++i) {
		// printf("\n");
		for (j=0; j < A.height; ++j) {
			// printf(" %3.2f ", A_elem[index(i, j, A.width)]);
			count++;
		}
		//printf("\n");
	}
	printf("\nContador = %d", count);

	// ahora la B
	count = 0;
	printf("\n\n -- Matriz B --\n");
	for (i=0; i< B.width ; ++i) {
		// printf("\n");
		for (j=0; j < B.height; ++j) {
			// printf(" %3.2f ", B_elem[index(i, j, B.width)]);
			count++;
		}
		//printf("\n");
	}
	printf("\nContador = %d", count);

	// Creación de las matrices en memoria de la GPU
	// Cálculo del tamaño de la matriz en memoria, matriz A
	size_t size = A.width*A.height*sizeof(double);
	A.elements = (double*)malloc(size);
	// Asignación elementos uno a uno
	for (i=0; i< A.width ; ++i) {
		for (j=0; j < A.height; ++j) {
			A.elements[i*A.width+j] = A_elem[index(i, j, A.width)];
		}
	}
	// Cálculo del tamaño de la matriz en memoria, matriz B
	size = B.width*B.height*sizeof(double);
	B.elements = (double*)malloc(size);
	for (i=0; i< B.width ; ++i) {
		for (j=0; j < B.height; ++j) {
			B.elements[i*B.width+j] = B_elem[index(i, j, B.width)];
		}
	}
	// pasar la mat a la funcion MulMat
	C.elements = (double*)malloc(size);
	// aca hay que incluir el codigo que lleva control del tiempo
	clock_t tinicio, t_GPU;
	float tg,tc;
	tinicio=clock();
	//
	MatMul(A,B,C);
	//
	t_GPU=clock();
	tg = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("\n\ntiempo de procesamiento (GPU): %6.3f s\n\n",tg);
	// aca se calculó el tiempo de la GPU, ahora la prueba y la CPU


	for (i=0; i< C.width ; ++i) {
		for (j=0; j < C.height; ++j) {
			C_elem_GPU[index(i, j, C.width)] = C.elements[i * C.width + j];
		}
	}
	printf("\n -- Matrix resultante (GPU) --\n");
	for (i=0; i< C.width ; ++i) {
		//printf("\n");
		for (j=0; j < C.height; ++j) {
			// printf(" %3.2f ", C_elem_GPU[index(i, j, C.width)]);
		}
		//printf("\n");
	}


	// aca vamos a realizar la multiplicacion de matrices mediante la cpu.
	// se analizaran los resultados.
	tinicio=clock();
	//
	for (i=0; i < A.height ; ++i) {
		for (j=0; j < B.width; ++j) {
			C_elem[index(i, j, C.width)] = 0;
			for(k=0 ; k < A.width; k++) {
				C_elem[index(i, j, C.width)] = C_elem[index(i, j, C.width)] + A_elem[index(i, k, A.width)] * B_elem[index(k, j, B.width)];
			}
		}
	}
	//
	t_GPU=clock();
	tc = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("\n\ntiempo de procesamiento (CPU): %6.3f s\n\n",tc);
	printf("\n -- Matrix resultante (CPU) - (GPU) --\n");
	for (i=0; i< C.width ; ++i) {
		printf("\n");
		for (j=0; j < C.height; ++j) {
			printf(" %3.2f -> %3.2f ", C_elem[index(i, j, C.width)], C.elements[index(i, j, C.width)]);
		}
		printf("\n");
	}
}
